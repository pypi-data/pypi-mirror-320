{# USES_VARIABLES { N } #}
{# ALLOWS_SCALAR_WRITE #}
{% extends 'common_group.cu' %}


{% block kernel_maincode %}
    ///// block kernel_maincode /////

    ///// scalar_code['condition'] /////
    {{scalar_code['condition']|autoindent}}

    ///// scalar_code['statement'] /////
    {{scalar_code['statement']|autoindent}}

    ///// vector_code['condition'] /////
    {{vector_code['condition']|autoindent}}

    if (_cond)
    {
        ///// vector_code['statement'] /////
        {{vector_code['statement']|autoindent}}
    }

    ///// endblock kernel_maincode /////
{% endblock kernel_maincode %}


{% block extra_kernel_call_post %}
    {# We need to copy modifed variables back to host in case they are used in
       codeobjects that run on the host, which are synapse connect calls (e.g. in the
       connect condition) and before run synapses push spikes, which initialized
       synaptic variables.
    #}
    {% for var, varname in written_variables.items() %}
    {% if var.dynamic %}
    {{varname}} = dev{{varname}};
    {% else %}
    CUDA_SAFE_CALL(
        hipMemcpy(
            {{varname}},
            dev{{varname}},
            sizeof({{c_data_type(var.dtype)}})*_num_{{varname}},
            hipMemcpyDeviceToHost
        )
    );
    {% endif %}
    {% endfor %}
{% endblock %}


{% block profiling_start %}
{% endblock %}


{% block profiling_stop %}
{% endblock %}
