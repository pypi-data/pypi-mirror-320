#include "hip/hip_runtime.h"
{# USES_VARIABLES { N, rate, t, _spikespace, _clock_t, _clock_dt,
                    _num_source_neurons, _source_start, _source_stop } #}
{# WRITES_TO_READ_ONLY_VARIABLES { N } #}
{% extends 'common_group.cu' %}

{% block define_N %}
{% endblock %}

{% block host_maincode %}
int current_iteration = {{owner.clock.name}}.timestep[0];
static int start_offset = current_iteration;
{% endblock %}

{% block prepare_kernel_inner %}
int num_iterations = {{owner.clock.name}}.i_end;
int size_till_now = dev{{_dynamic_t}}.size();
int new_size = num_iterations + size_till_now - start_offset;
THRUST_CHECK_ERROR(
        dev{{_dynamic_t}}.resize(new_size)
        );
THRUST_CHECK_ERROR(
        dev{{_dynamic_rate}}.resize(new_size)
        );
// Update size variables for Python side indexing to work
// (Note: Need to update device variable which will be copied to host in write_arrays())
_array_{{owner.name}}_N[0] = new_size;
CUDA_SAFE_CALL(
        hipMemcpy(dev_array_{{owner.name}}_N, _array_{{owner.name}}_N, sizeof(int32_t),
                   hipMemcpyHostToDevice)
        );

num_threads = 1;
num_blocks = 1;
{% endblock %}

{% block kernel_call %}
_run_kernel_{{codeobj_name}}<<<num_blocks, num_threads>>>(
    current_iteration - start_offset,
    thrust::raw_pointer_cast(&(dev{{_dynamic_rate}}[0])),
    thrust::raw_pointer_cast(&(dev{{_dynamic_t}}[0])),
    ///// HOST_PARAMETERS /////
    %HOST_PARAMETERS%);

CUDA_CHECK_ERROR("_run_kernel_{{codeobj_name}}");
{% endblock %}

{% block kernel %}
__global__ void
{% if launch_bounds %}
__launch_bounds__(1024, {{sm_multiplier}})
{% endif %}
_run_kernel_{{codeobj_name}}(
    int32_t current_iteration,
    {% set c_type = c_data_type(variables['rate'].dtype) %}
    {{c_type}}* ratemonitor_rate,
    {% set c_type = c_data_type(variables['t'].dtype) %}
    {{c_type}}* ratemonitor_t,
    ///// KERNEL_PARAMETERS /////
    %KERNEL_PARAMETERS%
    )
{
    using namespace brian;

    ///// KERNEL_CONSTANTS /////
    %KERNEL_CONSTANTS%

    ///// kernel_lines /////
    {{kernel_lines|autoindent}}

    int num_spikes = 0;

    if (_num_spikespace-1 != _num_source_neurons)  // we have a subgroup
    {
        // TODO shouldn't this be 'i < _num_spikespace -1'?
        for (int i=0; i < _num_spikespace; i++)
        {
            const int spiking_neuron = {{_spikespace}}[i];
            if (spiking_neuron != -1)
            {
                // check if spiking neuron is in this subgroup
                if (_source_start <= spiking_neuron && spiking_neuron < _source_stop)
                    num_spikes++;
            }
            else  // end of spiking neurons
            {
                break;
            }
        }
    }
    else  // we don't have a subgroup
    {
        num_spikes = {{_spikespace}}[_num_source_neurons];
    }

    // TODO: we should be able to use {{rate}} and {{t}} here instead of passing these
    //       additional pointers. But this results in thrust::system_error illegal memory access.
    //       Don't know why... {{rate}} and ratemonitor_rate should be the same...
    ratemonitor_rate[current_iteration] = 1.0*num_spikes/{{_clock_dt}}/_num_source_neurons;
    ratemonitor_t[current_iteration] = {{_clock_t}};
}
{% endblock %}
